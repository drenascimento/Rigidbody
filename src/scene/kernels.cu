#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_PARTICLES_IN_CELL 3
//#define SCAN_BLOCK_DIM 1024
//#include "exclusiveScan.cu_inl"

//#include <driver_functions.h>
//#include "../lib/mathlib.h"
//#include "../platform/gl.h"

__device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ float3 operator*(const float3&a, const float3 &b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

__device__ float3 operator*(const float& a, const float3 &b) {
    return make_float3(a*b.x, a*b.y, a*b.z);
}

__device__ float norm(const float3 &a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ float3 unit(const float3 &a) {
    float my_norm = norm(a);
    return make_float3(a.x/my_norm, a.y/my_norm, a.z/my_norm);
}

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 cross(const float3 &a, const float3 &b) {
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

__device__ float4 operator*(const float4 &a, const float4 &r) {
    return make_float4(a.y * r.z - a.z * r.y + a.x * r.w + a.w * r.x, a.z * r.x - a.x * r.z + a.y * r.w + a.w * r.y,
                           a.x * r.y - a.y * r.x + a.z * r.w + a.w * r.z, a.w * r.w - a.x * r.x - a.y * r.y - a.z * r.z);
}

__device__ float4 conjugate(const float4 &a) {
    return make_float4(-a.x, -a.y, -a.z, a.w);
}

__device__ float4 to4(const float3 &a) {
    return make_float4(a.x,a.y,a.z,0);
}

__device__ float3 rotate(const float4 &quat, const float3 &v) {
    float4 partial = (quat * to4(v)) * conjugate(quat);
    return make_float3(partial.x,partial.y,partial.z);
}

// Particles of the form [x,y,z,vx,vy,vz]
__global__ void step_kernel(int N, float *center_of_mass, float *quaternion, float *velocity, float *angular_velocity,
                            float* box_min, float* box_max, float *box_all, int *particle_indices, int *particle_owners, float particle_radius,
                            int num_particles, float *particles, int width, int height, int depth, float *out_force, float *out_torque, int *grid) {

    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    const int idy = threadIdx.y + blockDim.y * blockIdx.y;
    const int idz = threadIdx.z + blockDim.z * blockIdx.z;
    const int threadIndex = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    /* Clean grid */

    grid[threadIndex] = 0;
    __syncthreads();

    /* All threads < N are each responsible for 1 rigidbody. */

    if (threadIndex < N) {
        // Generate particles and store in local array
        float min_x = box_min[N * 3    ];
        float min_y = box_min[N * 3 + 1];
        float min_z = box_min[N * 3 + 2];

        float max_x = box_max[N * 3    ];
        float max_y = box_max[N * 3 + 1];
        float max_z = box_max[N * 3 + 2];

        const float r = particle_radius;
        const float start_x = - (max_x - min_x) / 2;
        const float end_x = -start_x;
        const float start_y = - (max_y - min_y) / 2;
        const float end_y = -start_y;
        const float start_z = - (max_z - min_z) / 2;
        const float end_z = -start_z;

        int i = particle_indices[threadIndex];
        for (float x = start_x + r; x <= end_x - r; x += 2*r) {
        for (float y = start_y + r; y <= end_y - r; y += 2*r) {
        for (float z = start_z + r; z <= end_z - r; z += 2*r) {
            /* Instantiate particle */
            float3 owner_center_of_mass = *((float3*)(center_of_mass + threadIndex * 3));
            float4 owner_quaternion = *((float4*)(quaternion + threadIndex * 4));
            float3 owner_vel = *((float3*)(velocity + threadIndex * 3));
            float3 owner_angular_vel = *((float3*)(angular_velocity + threadIndex * 3));

            float3 rel_pos = make_float3(x,y,z);

            float3 pos = owner_center_of_mass + rotate(owner_quaternion, rel_pos);
            float3 velocity = owner_vel + cross(owner_angular_vel, rotate(owner_quaternion, rel_pos));
            // Pos (x,y,z)
            particles[i * 6    ] = pos.x;
            particles[i * 6 + 1] = pos.y;
            particles[i * 6 + 2] = pos.z;
            // Vel (x,y,z)
            particles[i * 6 + 3] = velocity.x;
            particles[i * 6 + 4] = velocity.y;
            particles[i * 6 + 5] = velocity.z;

            /* Assign particle owner as rigid body */
            particle_owners[i] = threadIndex;

            /* Also store particles on grid */
            // No need to use a lock, if we miss a particle (who cares, right?) so long as they are sufficiently
            // small, their contribution will not be missed

            size_t x, y, z;
            x = floor((pos.x - box_all[0]) / (2.f * particle_radius));
            y = floor((pos.y - box_all[1]) / (2.f * particle_radius));
            z = floor((pos.z - box_all[2]) / (2.f * particle_radius));

            size_t index = (x * height * depth + y * depth + z) * MAX_PARTICLES_IN_CELL;

            int offset = 0;
            while (offset < MAX_PARTICLES_IN_CELL && grid[index + offset] != 0) offset++;

            if (offset < MAX_PARTICLES_IN_CELL) {
                grid[index + offset] = i;
            }

            i++;
        }}}

        /* Clean out_force and out_torque */
        out_force[threadIndex] = 0;
        out_torque[threadIndex] = 0;

        /* Debugging - DELETE */
        if (!(threadIndex == N-1 && i == num_particles) || (i == particle_indices[threadIndex+1])){
            printf("violation!\n");
        }
        /* Debugging - DELETE */
    }

    __syncthreads();

    /* Collision detection and reaction - write to out_force and out_torque */

    int index = threadIndex;
    int x = idx;
    int y = idy;
    int z = idz;

    for (size_t offset = 0; offset < MAX_PARTICLES_IN_CELL && grid[index + offset] != 0; offset++) {
        // Get all points in a 3x3 block around (x,y,z)
        for (int xp = x-1; xp <= x+1; xp++) {
        for (int yp = y-1; yp <= y+1; yp++) {
        for (int zp = z-1; zp <= z+1; zp++) {
            if (xp < 0 || xp >= width || yp < 0 || yp >= height || zp < 0 || zp >= depth) continue;

            size_t neighbor_index = (xp * height * depth + yp * depth + zp) * MAX_PARTICLES_IN_CELL;

            for (size_t neighbor_offset = 0; neighbor_offset < MAX_PARTICLES_IN_CELL && grid[neighbor_index + neighbor_offset] != 0; neighbor_offset++) {

                /* Possibly found a pair of colliding particles! Update my body only. */
                float3 my_particle_pos = *((float3*)particles + 6 * (grid[index + offset]));
                float3 neighbor_particle_pos = *((float3*)particles + 6 * (grid[neighbor_index + neighbor_offset]));

                float3 my_particle_velocity = *((float3*)particles + 6 * (grid[index + offset]) + 3);
                float3 neighbor_particle_velocity = *((float3*)particles + 6 * (grid[neighbor_index + neighbor_offset]) + 3);

                if (particle_owners[grid[index + offset]] == particle_owners[grid[neighbor_index + neighbor_offset]]) continue;

                float real_spring_coefficient = 0.5f;
                float real_shear_coefficient = 0.5f;
                float real_damping_coefficient = 0.5f;

                // We want the relative pos/vel of neighbor with respect to ourselves
                float3 rel_pos_other = neighbor_particle_pos - my_particle_pos;
                float3 rel_vel_other = neighbor_particle_velocity - my_particle_velocity;
                float3 rel_tangential_vel = rel_vel_other - (dot(rel_vel_other,unit(rel_pos_other)) * unit(rel_pos_other));

                float3 Fis = -real_spring_coefficient * abs(2.f * particle_radius - norm(rel_pos_other)) * unit(rel_pos_other);
                float3 Fid = real_damping_coefficient * rel_vel_other;
                float3 Fit = real_shear_coefficient * rel_tangential_vel;

                // Write updates to out_force and out_torque. For this we will need synchronization primitives

                float3 center_of_mass_owner = *((float3 *)center_of_mass + 3 * particle_owners[grid[index + offset]]);
                float3 rel_pos_to_center = my_particle_pos - center_of_mass_owner;

                float3 total_force = Fis + Fid + Fit;
                float3 total_torque = cross(rel_pos_to_center, total_force);

                atomicAdd(out_force + 3 * (particle_owners[grid[index + offset]]    ), total_force.x);
                atomicAdd(out_force + 3 * (particle_owners[grid[index + offset]] + 1), total_force.y);
                atomicAdd(out_force + 3 * (particle_owners[grid[index + offset]] + 2), total_force.z);

                atomicAdd(out_torque + 3 * (particle_owners[grid[index + offset]]    ), total_torque.x);
                atomicAdd(out_torque + 3 * (particle_owners[grid[index + offset]] + 1), total_torque.y);
                atomicAdd(out_torque + 3 * (particle_owners[grid[index + offset]] + 2), total_torque.z);
            }
        }}}
    }

}

/* In:
 * Center of mass, size 3N
 * Quaternion, size 4N
 * Box min, size 3N
 * Box max, size 3N
 * Box all, size 6 - (x,y,z) min, (x,y,z) max
 *
 * Out:
 * outForce, size N
 * outTorque, size N */
void
update(int N, float particle_radius, float *center_of_mass, float *quaternion, float *velocity, float* angular_velocity,
       float* box_min, float* box_max, float *box_all, float *out_force, float *out_torque) {

    // compute number of blocks and threads per block
    int width  = ceilf((box_all[3] - box_all[0]) / (2.f * particle_radius));
    int height = ceilf((box_all[4] - box_all[1]) / (2.f * particle_radius));
    int depth  = ceilf((box_all[5] - box_all[2]) / (2.f * particle_radius));

    const int block_x = 16;
    const int block_y = 8;
    const int block_z = 8;

    dim3 block(block_x, block_y, block_z);
    dim3 grid((width + block_x - 1) / block_x, (height + block_y - 1) / block_y, (depth + block_z - 1) / block_z);

    int particle_indices[N];
    int num_particles = 0;
    particle_indices[0] = 0;

    for (int i=0; i<N; i++) {
        // Get particle count of ith rigid body
        float min_x = box_min[N * 3    ];
        float min_y = box_min[N * 3 + 1];
        float min_z = box_min[N * 3 + 2];

        float max_x = box_max[N * 3    ];
        float max_y = box_max[N * 3 + 1];
        float max_z = box_max[N * 3 + 2];

        const float r = particle_radius;
        const float start_x = - (max_x - min_x) / 2;
        const float end_x = -start_x;
        const float start_y = - (max_y - min_y) / 2;
        const float end_y = -start_y;
        const float start_z = - (max_z - min_z) / 2;
        const float end_z = -start_z;

        int num_particles_x = floor((end_x - r) - (start_x + r)) / (2*r) + 1;
        int num_particles_y = floor((end_y - r) - (start_y + r)) / (2*r) + 1;
        int num_particles_z = floor((end_z - r) - (start_z + r)) / (2*r) + 1;

        if (i == N-1) {
            num_particles = num_particles_x * num_particles_y * num_particles_z + particle_indices[i];
        } else {
            particle_indices[i+1] = num_particles_x * num_particles_y * num_particles_z + particle_indices[i];
        }
    }

    //const int threadsPerBlock = 512;
    //const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_center_of_mass;
    float* device_quaternion;
    float* device_box_min;
    float* device_box_max;
    float* device_box_all;
    float* device_velocity;
    float* device_angular_velocity;
    int* device_particle_indices;
    float* device_particles;
    int* device_particle_owners;
    int* device_grid;
    float* device_out_force;
    float* device_out_torque;

    hipMalloc((void **) &device_center_of_mass, N * 3 * sizeof(float));
    hipMalloc((void **) &device_quaternion, N * 4 * sizeof(float));
    hipMalloc((void **) &device_velocity, N * 3 * sizeof(float));
    hipMalloc((void **) &device_angular_velocity, N * 3 * sizeof(float));
    hipMalloc((void **) &device_box_min, N * 3 * sizeof(float));
    hipMalloc((void **) &device_box_max, N * 3 * sizeof(float));
    hipMalloc((void **) &device_box_all, 6 * sizeof(float));
    hipMalloc((void **) &device_particle_indices, N * sizeof(int));
    hipMalloc((void **) &device_particles, num_particles * 6 * sizeof(float));
    hipMalloc((void **) &device_out_force, N * 3 * sizeof(float));
    hipMalloc((void **) &device_out_torque, N * 3 * sizeof(float));
    hipMalloc((void **) &device_grid, width * height * depth * 3 * sizeof(int));
    hipMalloc((void **) &device_particle_owners, num_particles * sizeof(int));

    hipMemcpy(device_center_of_mass, center_of_mass, N * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_quaternion, quaternion, N * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_box_min, box_min, N * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_box_max, box_max, N * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_box_all, box_all, 6 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_particle_indices, particle_indices, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_velocity, velocity, N * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_angular_velocity, angular_velocity, N * 3 * sizeof(float), hipMemcpyHostToDevice);

    step_kernel<<<grid, block>>>(N, device_center_of_mass, device_quaternion, device_velocity, device_angular_velocity, device_box_min,
                                 device_box_max, device_box_all, device_particle_indices, device_particle_owners, particle_radius,
                                 num_particles, device_particles, width, height, depth, device_out_force, device_out_torque, device_grid);

    hipDeviceSynchronize();

    hipMemcpy(out_force, device_out_force, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(out_torque, device_out_torque, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_center_of_mass);
    hipFree(device_quaternion);
    hipFree(device_box_min);
    hipFree(device_box_max);
    hipFree(device_velocity);
    hipFree(device_angular_velocity);
    hipFree(device_particles);
    hipFree(device_particle_indices);
    hipFree(device_out_force);
    hipFree(device_out_torque);
    hipFree(device_grid)
}
